#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

inline double gettime_ms() {
        struct timeval t;
        gettimeofday(&t,NULL);
        return (t.tv_sec+t.tv_usec*1e-6)*1000;
}

__global__ void mem_kernel(int *arr, int N, int *rdom_arr, int *result) { 
  int tem=0;
  int random;
  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int totalthreads=blockDim.x*gridDim.x;

  for(int i=0;i<10;++i){

    for(int j=tid ;j<N; j+=totalthreads){
      random = rdom_arr[j];
      int idx=(j+random)%N;
      tem += arr[idx];
      result[random%N]=tem;    
    } 
  
  } 
  //write result
}

int main(int argc, char **argv){

//input: Size N, Utilization U, Affinity A, BlockNum B, ThreadsNum T, Randomizer X
int N, U, A, B, T, X;
if (argc!=7) {
	printf("\nInput arguments wrong!\n input: Size N, Utilization U, Affinity A, BlockNum B, ThreadsNum T, Random X  \n ");
	return 0;
}
N=atoi(argv[1]);
U=atof(argv[2]);
A=atoi(argv[3]);
B=atoi(argv[4]);
T=atoi(argv[5]);
X=atoi(argv[6]);


printf("\ninput: Size N:%d, Utilization U:%d, Affinity A:%d, BlockNum B:%d, ThreadsNum T:%d, Random X:%d  \n ",N,U,A,B,T,X);

if(N%32!=0) {
	printf("\nArray size N has to be multiple of 32\n");
	return 0;
}

hipSetDevice(0);
srand(0);

int *array_h=(int *)malloc(N*sizeof(int));
for (int i=0;i<N;++i) array_h[i]=1;
int *array_d;
hipMalloc(&array_d,N*sizeof(int));
hipMemcpy(array_d,array_h,N*sizeof(int),hipMemcpyHostToDevice);

int *rdom_arr_h=(int *)malloc(N*sizeof(int));
for (int i=0;i<N;++i) rdom_arr_h[i]= rand() % X; //generate random number in range [0, X)
int *rdom_arr_d;
hipMalloc(&rdom_arr_d,N*sizeof(int));
hipMemcpy(rdom_arr_d,rdom_arr_h,N*sizeof(int),hipMemcpyHostToDevice);

int *result_h=(int *)malloc(N*sizeof(int));
int *result_d;
hipMalloc(&result_d,N*sizeof(int));

double ktime=gettime_ms();
mem_kernel<<<B,T>>>(array_d,N,rdom_arr_d,result_d);
hipDeviceSynchronize();
ktime=gettime_ms()-ktime;
FILE* fp=fopen("gpu_result.txt","a+");
fprintf(fp,"%f ",ktime);
printf("Kernel time:%f \n",ktime);
fclose(fp);
hipMemcpy(result_h,result_d,N*sizeof(int),hipMemcpyDeviceToHost);
hipDeviceSynchronize();

printf("results:\n");
for(int i=0;i<10;++i){
	printf("%d ",result_h[i]);
}
printf("\n");
free(array_h);
free(rdom_arr_h);
free(result_h);
hipFree(array_d);
hipFree(rdom_arr_d);
hipFree(result_d);
return 0;
}
